#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <iostream>

/* --------------------------- target code ------------------------------*/  

// hipArray* array;
// texture<float,2,hipReadModeNormalizedFloat> texRef;
 
__global__ void copy_image_impl (uint8_t* in_data, int width, int height) 
{ 
  //  hipBindTextureToArray(texRef, array);

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i < width && j < height) {
    int base_index = 4 * (j * width + i);
    // out_data[base_index] = in_data[base_index];
    // out_data[base_index+1] = in_data[base_index+1];
    // out_data[base_index+2] = in_data[base_index+2];
    // out_data[base_index+3] = in_data[base_index+3];
    in_data[base_index] = 0;//in_data[base_index];
    in_data[base_index+1] = 0;
    in_data[base_index+2] = in_data[base_index+2];
    in_data[base_index+3] = 255;
  }
} 
 
/* --------------------------- host code ------------------------------*/ 

void copy_image (int pbo_in, int width, int height) {

  uint8_t* in_data;
  cudaGLMapBufferObject( (void**)&in_data, pbo_in );

  // hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
  // cudaGLMapBufferObject( (void**)&array, pbo_in );
  // hipBindTextureToArray(texRef, array, channelDesc);

  std::cout << "grid size: " << width << "  " << height << "\n";
  dim3 block(16,16);
  dim3 grid((width + block.x - 1)  / block.x,
            (height + block.y - 1) / block.y);
  std::cout << "grid size: " << grid.x << "  " << grid.y << "\n";
  copy_image_impl<<<grid,block>>>(in_data, width, height); 
  
  cudaGLUnmapBufferObject( pbo_in );
} 

void pboRegister(int pbo) {
  // register this buffer object with CUDA
  cudaGLRegisterBufferObject(pbo);
}
void pboUnregister(int pbo) {
  // unregister this buffer object with CUDA
  cudaGLUnregisterBufferObject(pbo);
}
